
#include <hip/hip_runtime.h>
#include <string>
#include <iostream>
#include <sstream>
#include <fstream>
#include "stdio.h"
#include "stdlib.h"
#include "math.h"
#include "time.h"


#define Nx			510		    // Dimension X
#define Ny			510			// Dimension Y
#define Nz			510			// Dimension Z

//Mapping function
#define STRIDE			((Ny+2)*(Nz+2))
#define WIDTH			(Nz+2)
#define pos(x,y,z)		(STRIDE*(x)+WIDTH*(y)+(z))

#define LENMAX		256
#define REAL		double

////////////////////////////
// CPU I/O function
////////////////////////////
void Init(REAL *P1,REAL *U1);
void WriteFields(REAL *P,REAL *U);


////////////////////////////////////////////////
//              Main CPU program              //
////////////////////////////////////////////////
int main(int argc, char **argv)
{

	size_t SizeGrid = (Nx+2)*(Ny+2)*(Nz+2);
	REAL *h_Psi=(REAL*)malloc(SizeGrid*sizeof(REAL)) ;	
	REAL *h_U=(REAL*)malloc(SizeGrid*sizeof(REAL)) ;	
	

	clock_t begin=clock();

    printf("Initializing...\n");
	Init(h_Psi,h_U);
	WriteFields(h_Psi,h_U);
	
	clock_t end=clock();
	REAL CompTime=(end-begin)/CLOCKS_PER_SEC;
    printf("\n\nThe time of writing binary files (2 GB) is %d s\n", int(CompTime));
    

	free(h_Psi) ;
	free(h_U) ;
	
	return EXIT_SUCCESS;
}


/////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////// Initialization //////////////////////////////
/////////////////////////////////////////////////////////////////////////////////
void Init(REAL *P1,REAL *U1)
{
	REAL r=((REAL) rand() / (RAND_MAX));
	for (int i=0;i<Nx+2;i++) {
		for (int j=0;j<Ny+2;j++) {
			for (int k=0;k<Nz+2;k++) {
				    
					P1[pos(i,j,k)]=r;
					U1[pos(i,j,k)]=r*r;	
			}
		}
	}
}


/////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////// Output /////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////
void WriteFields(REAL *P,REAL *U)
{
	//================================================
	// Output dat file
	//================================================
	char FileName1[256];
	FILE *OutFile1;	
    sprintf(FileName1,"File1.dat");
    OutFile1=fopen(FileName1,"w");
    
    for(int i=0; i<Nx+2; i++) {
        for(int j=0; j<Ny+2; j++) {
            for(int k=0; k<Nz+2; k++) {
                
                REAL d = P[pos(i,j,k)];
                
                fwrite((char*)&d,sizeof(REAL),1,OutFile1);
                
            }
        }
    }
    fclose(OutFile1);
    printf("Written File1.dat \n");	
	
	char FileName2[256];
	FILE *OutFile2;	
    sprintf(FileName2,"File2.dat");
    OutFile2=fopen(FileName2,"w");
    
    for(int i=0; i<Nx+2; i++) {
        for(int j=0; j<Ny+2; j++) {
            for(int k=0; k<Nz+2; k++) {
                
                REAL d = U[pos(i,j,k)];
                
                fwrite((char*)&d,sizeof(REAL),1,OutFile2);
                
            }
        }
    }
    fclose(OutFile2);
	printf("Written File2.dat \n");	    
}